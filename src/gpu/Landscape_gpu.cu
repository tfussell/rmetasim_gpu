#include "hip/hip_runtime.h"

/*
rmetasim_gpu: A CUDA implementation of R package "rmetasim"
Copyright (C) 2012 Thomas Fussell

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/* External Includes */
#include <cassert>
#include <ctime>
#include <thrust/count.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>
#include <thrust/binary_search.h>

/* Local Includes */
#include "gpu/Landscape_gpu.cuh"
#include "gpu/const_gpu.h"
#include "gpu/CudaKernels.cuh"
#include "gpu/ThrustFunctors.h"
#include "gpu/CudaIndividual.h"
#include "gpu/PRNG.h"

namespace rmetasim_gpu {

// Constructors/Destructors

Landscape_gpu::Landscape_gpu() :
    timer_(),
    rand_seed_(0),
    initialized_(false),
    num_habitats_(0),
    num_stages_(0),
    num_classes_(0),
    max_size_(0),
    carrying_capacity_(0),
    multiple_paternity_(false),
    selfing_rate_(0.0),
    sum_loci(0),
    current_generation_(0),
    next_individual_id_(0)
{

}

Landscape_gpu::~Landscape_gpu()
{

}

// Host<=>Device Memory Transfer

void Landscape_gpu::FromLandscape(const unsigned int rand_seed, Landscape_statistics &landscape)
{
    timer_.BeginMethod("Initialize");

    num_habitats_ = landscape.gethabs();
    num_stages_ = landscape.getstages();
    num_classes_ = num_habitats_ * num_stages_;
    multiple_paternity_ = landscape.getmultp();
    selfing_rate_ = landscape.getself();
    current_generation_ = landscape.getCgen();
    next_individual_id_ = landscape.getnextID();

    habitat_extinction_rates_.resize(num_habitats_);
    habitat_carrying_capacities_.resize(num_habitats_);

    landscape.getextinct(0, &habitat_extinction_rates_[0]);
    landscape.getk(0, &habitat_carrying_capacities_[0]);

    host_S_.resize(num_classes_ * num_classes_);
    host_R_.resize(num_classes_ * num_classes_);
    host_M_.resize(num_classes_ * num_classes_);

    S_.resize(num_classes_ * num_classes_);
    R_.resize(num_classes_ * num_classes_);
    M_.resize(num_classes_ * num_classes_);    

    for(int from = 0; from < num_classes_; from++)
    {
	for(int to = 0; to < num_classes_; to++)
	{
	    //XXX: should this be from*num_classes or to*num_classes?
	    int linear_index = from * num_classes_ + to;
	    
	    host_S_[linear_index] = landscape.getSmatElement(0, to, from);
	    host_R_[linear_index] = landscape.getRmatElement(0, to, from);
	    host_M_[linear_index] = landscape.getMmatElement(0, to, from);
	}
    }

    SetTransitionMatrices();

    std::vector<CudaIndividual> individuals;
    ExtractIndividualsFromLandscape(landscape, individuals);
	
    CalculateCapacities();
    ResizeVectors();

    if(individuals.size() > 0)
    {
	assert(individuals.size() <= max_size_);
	SetIndividuals(individuals);
    }

    InitializeKernelParameters();
    InitializeRandStates();
    initialized_ = true;

    timer_.EndMethod();
}

void Landscape_gpu::ToLandscape(Landscape_statistics &landscape)
{
    timer_.BeginMethod("Terminate");

    std::list<CudaIndividual> individuals;
    GetIndividuals(individuals);
    InjectIndividualsIntoLandscape(individuals, landscape);

    initialized_ = false;

    timer_.EndMethod();
}

// Simulation Interface

void Landscape_gpu::Simulate(const int num_iterations, const bool compress, const int by_population)
{
//    printf("\nBeginning Simulation\n");

    for(int i = 0; i < num_iterations; i++)
    {
//	printf("Step %d\n", i);

	const int pop_before = CountIndividuals();

	if(pop_before > 0)
	{
	    Extirpate();
	    ReproduceAndSurvive();
	    LambdaAdjust(by_population);
	    LandCarry();
	    HabCarry();
	}

	Advance();

//	int pop_after = CountIndividuals();
//	int pop_delta = pop_after - pop_before;

//	printf("Population Sizes: before=%d after=%d change=%d\n", pop_before, pop_after, pop_delta);
//	printf("Step %d End\n", i);
    }

    if(compress)
    {
	Survive();
    }

    LandCarry();
    HabCarry();

//    printf("\nSimulation Completed\n");

//    PrintSimulationStatistics();
}

int Landscape_gpu::PopSize(const int habitat)
{
    return habitat >= 0 ? CountIndividualsInHabitat(habitat) : CountIndividuals();
}

void Landscape_gpu::Extirpate()
{
    timer_.BeginMethod("Extirpate");

    bool changed = false;

    for(int habitat_index = 0; habitat_index < num_habitats_; habitat_index++)
    {
	double rn = rand() / (double)RAND_MAX;

	if(habitat_extinction_rates_[habitat_index] > rn)
	{
	    changed = true;

	    int min_class = habitat_index * num_stages_;
	    int max_class = min_class + num_stages_;

	    thrust::transform(classes_.begin(), classes_.end(), classes_.begin(), functors::kill_in_range(min_class, max_class));
	}
    }

    if(changed)
    {
	CompactIndividuals();
    }

    timer_.EndMethod();
}

void Landscape_gpu::ReproduceAndSurvive()
{
    timer_.BeginMethod("ReproduceAndSurvive");

    int current_pop_size = CountIndividuals();
    int num_parents = current_pop_size;

    CalculateMaleGameteMatrix();
	
    for(int to = 0; to < num_classes_; to++)
    {
	thrust::fill(num_offspring_.begin(), num_offspring_.begin() + current_pop_size, 0);

	kernel_params_.to_state = to;
	kernel_params_.num_individuals = current_pop_size;
	
	cuda::CalculateRandomNumberOfOffspring(kernel_params_);
	
	int num_offspring = thrust::count_if(num_offspring_.begin(), 
	    num_offspring_.begin() + current_pop_size, functors::positive_int());
		
	thrust::replace_if(generations_last_reproduced_.begin(), 
	    generations_last_reproduced_.end(), num_offspring_.begin(), 
	    functors::positive(), current_generation_);   

//	printf("Reproduce(): to=%d num_offspring=%d\n",to,num_offspring);

	if(num_offspring > 0)
	{
	    int num_chunks = (num_offspring - 1) / OFFSPRING_ALLOC_CHUNK_SIZE + 1;
	    int remaining_offspring = num_offspring;

//	    printf("Using %d chunks for offspring\n",num_chunks); 
    
	    for(int i = 0; i < num_chunks; i++)
	    {
		int chunk_size = std::min(OFFSPRING_ALLOC_CHUNK_SIZE, remaining_offspring);
		int chunk_start_index = i * OFFSPRING_ALLOC_CHUNK_SIZE;	

//		printf("Chunk %d: offspring from indices %d to %d\n", i, chunk_start_index, chunk_start_index + chunk_size);

		//Find Parents
		thrust::fill(offspring_maternal_indices_.begin(),
		    offspring_maternal_indices_.end(), -1);
		thrust::fill(offspring_paternal_indices_.begin(),
		    offspring_paternal_indices_.end(), -1);	    
		thrust::fill(offspring_classes_.begin(),
		    offspring_classes_.end(), -1);

		thrust::fill(offspring_classes_.begin(),
		    offspring_classes_.begin() + chunk_size, to);
	    
		thrust::inclusive_scan(num_offspring_.begin(), num_offspring_.end(), 
		    num_offspring_.begin());

		thrust::lower_bound(num_offspring_.begin(), num_offspring_.end(), 
		    thrust::counting_iterator<int>(chunk_start_index), 
		    thrust::counting_iterator<int>(chunk_start_index + chunk_size), 
		    offspring_maternal_indices_.begin());

		kernel_params_.to_state = to;
		kernel_params_.num_individuals = chunk_size;

		//cuda::FindMates(kernel_params_);
		// End Find Parents

		cuda::SurviveOffspring(kernel_params_);

		int num_living_offspring_in_chunk = thrust::count_if(
		    offspring_classes_.begin(), offspring_classes_.end(), 
		    functors::non_negative());

//		printf("%d offspring remaining in chunk after survive.\n", num_living_offspring_in_chunk);

		// Stream compaction of offspring vectors
		thrust::copy_if(
		    thrust::make_zip_iterator(
			thrust::make_tuple(
			    offspring_maternal_indices_.begin(),
			    offspring_paternal_indices_.begin(),
			    offspring_classes_.begin())),
		    thrust::make_zip_iterator(
			thrust::make_tuple(
			    offspring_maternal_indices_.end(),
			    offspring_paternal_indices_.end(),
			    offspring_classes_.end())),
		    offspring_classes_.begin(),
		    thrust::make_zip_iterator(
			thrust::make_tuple(
			    offspring_maternal_indices_.begin(),
			    offspring_paternal_indices_.begin(),
			    offspring_classes_.begin())),
		    functors::non_negative());

		//Offspring Block
		{
		    //cuda::SexualReproduction(kernel_params_);
		}
	
		// Transfer living offspring to real landscape
		if(num_living_offspring_in_chunk > 0)
		{
		    assert(current_pop_size + num_living_offspring_in_chunk <= max_size_);
		    
		    if(next_individual_id_ + num_living_offspring_in_chunk < MAXIDS)
		    {
			thrust::copy(thrust::make_counting_iterator(next_individual_id_), 
			    thrust::make_counting_iterator(next_individual_id_ + num_living_offspring_in_chunk), 
			    ids_.begin() + current_pop_size);
			next_individual_id_ += num_living_offspring_in_chunk;
		    }
		    else
		    {
			// ID wraparound occured; Assign ids in two sets.
			// First set is range [next_individual_id_,MAXIDS).
			// Second set is in range [1, ((next_individual_id_ + num_offspring) - MAXIDS)).
			int under = MAXIDS - next_individual_id_;
			int over = (next_individual_id_ + num_living_offspring_in_chunk) - MAXIDS;
			
//			printf("ID wraparound: next=%d num_offspring=%d under=%d over=%d\n",next_individual_id_,num_living_offspring_in_chunk,under,over);
		
			thrust::copy(thrust::make_counting_iterator(next_individual_id_), 
			    thrust::make_counting_iterator(MAXIDS), 
			    ids_.begin() + current_pop_size);
			thrust::copy(thrust::make_counting_iterator(1), 
			    thrust::make_counting_iterator(over), 
			    ids_.begin() + current_pop_size + under);
		
			next_individual_id_ = over;
		    }
	    
		    thrust::copy(offspring_classes_.begin(),
			offspring_classes_.begin() + num_living_offspring_in_chunk,
			classes_.begin() + current_pop_size);

		    thrust::copy(offspring_genotypes_.begin(),
			offspring_genotypes_.begin() + num_living_offspring_in_chunk,
			genotypes_.begin() + current_pop_size);
	    
		    thrust::fill(generations_born_.begin() + current_pop_size, 
			generations_born_.begin() + current_pop_size + num_living_offspring_in_chunk,
			current_generation_);
	    
		    thrust::fill(generations_last_changed_.begin() + current_pop_size, 
			generations_last_changed_.begin() + current_pop_size + num_living_offspring_in_chunk,
			current_generation_);
	    
		    //ReproduceInitializeNewbornOffspring();

		    current_pop_size += num_living_offspring_in_chunk;

		    // Remove this in production code.
		    // Ensures that reproduction is functioning correctly.
		    int check_pop_size = CountIndividuals();
		    assert(current_pop_size == check_pop_size);
		}
	    }
	}
    }

    //Survive Parents Only
    {
	kernel_params_.num_individuals = num_parents;
	kernel_params_.current_generation = current_generation_;
	
	cuda::Survive(kernel_params_);
	
	CompactIndividuals();
    }

    timer_.EndMethod();
}

void Landscape_gpu::Survive()
{
    timer_.BeginMethod("Survive");

    kernel_params_.current_generation = current_generation_;

    cuda::Survive(kernel_params_);
    CompactIndividuals();

    timer_.EndMethod();
}


void Landscape_gpu::LambdaAdjust(const int by_population)
{
    timer_.BeginMethod("LambdaAdjust");

    int i, j, k, l, bigto, bigfrom;
    double pred_l, sim_l, adjrate;
    TransMat diag, Spopmat, Rpopmat;

    if(by_population != 0)
    {
	if(by_population == 1) 
	{
	    diag.SetSize(num_stages_);
	    Spopmat.SetSize(num_stages_);
	    Rpopmat.SetSize(num_stages_);

	    for(i = 0; i < num_habitats_; i++)
	    {
		for(j = 0; j < num_stages_; j++)
		{
		    for(k = 0; k < num_stages_; k++)
		    {
			bigto = (i * num_stages_) + k;
			bigfrom = (i * num_stages_) + j;

			Spopmat.SetElement(k, j, host_S_[num_classes_ * bigto + bigfrom]);
			Rpopmat.SetElement(k, j, host_R_[num_classes_ * bigto +  bigfrom]);
		    }
		}

		pred_l = (Spopmat + Rpopmat).Lambda();
		sim_l = (Spopmat * (Rpopmat + diag)).Lambda();
		adjrate = pred_l / sim_l;

		for(l = (i * num_stages_); l < ((i * num_stages_) + num_stages_); l++)
		{
		    CarryState(int(round(double(CountIndividualsInClass(l)) * adjrate)), l);
		}
	    }
	}
	else
	{
	    assert(1 == 0);
/*
	    diag.SetSize(num_classes_);
	    pred_l = (S[e]+R[e]).Lambda();
	    sim_l = (S[e]*(R[e]+diag)).Lambda();
	    adjrate = pred_l/sim_l;
	    
	    for (i=0;i<(s*nhab);i++)
	    {
		CarryState(int(round(double(I[i].size())*adjrate)),i);
	    }
*/
	} // if(by_population == 1)
    } // if(by_population != 0)

    timer_.EndMethod();
}

void Landscape_gpu::HabCarry(const int k)
{
    std::vector<double> prop(num_habitats_);

    for(int h = 0; h < num_habitats_; h++)
    {
	if (k < 0)
	{
	    prop[h] = double(habitat_carrying_capacities_[h]) / double(CountIndividualsInHabitat(h));
	}
	else
	{
	    prop[h] = double(k) / double(CountIndividualsInHabitat(h));
	}

	if (prop[h] > 1) 
	{
	    prop[h] = 1.0;
	}
    }

    for(int j = 0; j < num_classes_; j++)
    {
	int habitat = j / num_stages_;
	CarryState(prop[habitat] * CountIndividualsInClass(j), j);
    }
}

void Landscape_gpu::LandCarry()
{
    const int pop_size = CountIndividuals();
    const double pr = static_cast<double>(carrying_capacity_) / pop_size;

    for (int j = 0; j < num_classes_; j++)
    {
	CarryState(pr * CountIndividualsInClass(j), j);
    }
}

void Landscape_gpu::CarryState(const int max_size, const int state)
{
    timer_.BeginMethod("CarryState");

    const int total_pop_size = CountIndividuals();
    const int state_count = CountIndividualsInClass(state);

    if (max_size < state_count)
    {
	//XXX:remove this line, just for testing
//	CompactIndividuals();

	int num_to_delete = state_count - max_size;

	thrust::fill(shuffling_dartboard_.begin(), shuffling_dartboard_.end(),
	    -1);

	kernel_params_.state = state;
	kernel_params_.num_individuals = total_pop_size;

	cuda::FillShuffleVector(kernel_params_);

	const int state_size_check = thrust::count_if(
	    shuffling_dartboard_.begin(), shuffling_dartboard_.end(), 
	    functors::non_negative_int());

	assert(state_size_check == state_count);

	thrust::copy_if(shuffling_dartboard_.begin(),
	    shuffling_dartboard_.end(), shuffling_dartboard_.begin(), 
	    functors::non_negative());

	thrust::fill(shuffling_dartboard_.begin() + num_to_delete,
	    shuffling_dartboard_.end(), -1);

	thrust::fill(
	    thrust::make_permutation_iterator(
		classes_.begin(), 
		shuffling_dartboard_.begin()),
	    thrust::make_permutation_iterator(
		classes_.begin() + num_to_delete, 
		shuffling_dartboard_.begin() + num_to_delete), -1);

	//const int pop_size_after = CountIndividualsInClass(state);
	//assert(pop_size_after == max_size);

	CompactIndividuals();
    }

    timer_.EndMethod();
}

void Landscape_gpu::Advance()
{
    current_generation_++;
}

// Initialization

void Landscape_gpu::CalculateCapacities()
{
    carrying_capacity_ = 0;

    for(int i = 0; i < num_habitats_; i++)
    {
	carrying_capacity_ += habitat_carrying_capacities_[i];
    }

    max_size_ = carrying_capacity_ * LANDSCAPE_VECTOR_SIZE_MULTIPLIER;
}

void Landscape_gpu::InitializeKernelParameters()
{
    // Landscape parameters
    kernel_params_.max_individuals = max_size_;
    kernel_params_.num_classes = num_classes_;
    kernel_params_.num_habitats = num_habitats_;
    kernel_params_.num_stages = num_stages_;
    kernel_params_.selfing_rate = selfing_rate_;
    kernel_params_.multiple_paternity = multiple_paternity_;

    // Individual vectors
    kernel_params_.generations_born = thrust::raw_pointer_cast(
	generations_born_.data());
    kernel_params_.generations_last_changed = thrust::raw_pointer_cast(
	generations_last_changed_.data());
    kernel_params_.generations_last_reproduced = thrust::raw_pointer_cast(
	generations_last_reproduced_.data());
    kernel_params_.classes = thrust::raw_pointer_cast(classes_.data());
    kernel_params_.ids = thrust::raw_pointer_cast(ids_.data());
    kernel_params_.maternal_ids = thrust::raw_pointer_cast(
	maternal_ids_.data());
    kernel_params_.paternal_ids = thrust::raw_pointer_cast(
	paternal_ids_.data());
    kernel_params_.num_offspring = thrust::raw_pointer_cast(
	num_offspring_.data());
    kernel_params_.genotypes = thrust::raw_pointer_cast(
	genotypes_.data());

    // Shuffling vectors
    kernel_params_.shuffling_dartboard = thrust::raw_pointer_cast(
	shuffling_dartboard_.data());

    // Offspring vectors
    kernel_params_.offspring_maternal_indices = thrust::raw_pointer_cast(
	offspring_maternal_indices_.data());
    kernel_params_.offspring_paternal_indices = thrust::raw_pointer_cast(
	offspring_paternal_indices_.data());
    kernel_params_.offspring_classes = thrust::raw_pointer_cast(
	offspring_classes_.data());
    kernel_params_.offspring_genotypes = thrust::raw_pointer_cast(
	offspring_genotypes_.data());

    // Transition matrices
    kernel_params_.S = thrust::raw_pointer_cast(S_.data());
    kernel_params_.R = thrust::raw_pointer_cast(R_.data());
    kernel_params_.M = thrust::raw_pointer_cast(M_.data());

    // RNG state vector
    kernel_params_.rand_states = thrust::raw_pointer_cast(
	rand_states_.data());
}

void Landscape_gpu::InitializeRandStates()
{
    PRNG rng(rand_seed_);
    thrust::host_vector<unsigned long> h_per_thread_seeds(max_size_);
	
    for(int i = 0; i < max_size_; i++)
    {
	h_per_thread_seeds[i] = rng.RandomUInt();
    }

    thrust::device_vector<unsigned long> d_per_thread_seed = h_per_thread_seeds;
    cuda::InitializeRandStates(thrust::raw_pointer_cast(d_per_thread_seed.data()), kernel_params_);
}

void Landscape_gpu::ResizeVectors()
{   
    generations_born_.resize(max_size_);
    generations_last_changed_.resize(max_size_);
    generations_last_reproduced_.resize(max_size_);
    classes_.resize(max_size_);
    ids_.resize(max_size_);
    maternal_ids_.resize(max_size_);
    paternal_ids_.resize(max_size_);
    num_offspring_.resize(max_size_);
    genotypes_.resize(max_size_);

    shuffling_dartboard_.resize(max_size_);

    offspring_maternal_indices_.resize(OFFSPRING_ALLOC_CHUNK_SIZE);
    offspring_paternal_indices_.resize(OFFSPRING_ALLOC_CHUNK_SIZE);
    offspring_classes_.resize(OFFSPRING_ALLOC_CHUNK_SIZE);
    offspring_genotypes_.resize(OFFSPRING_ALLOC_CHUNK_SIZE);

    rand_states_.resize(max_size_);
}

// Landscape_gpu<=>Landscape Individuals Transfer

void Landscape_gpu::ExtractIndividualsFromLandscape(Landscape_statistics &landscape, std::vector<CudaIndividual> &individuals)
{
    for(int demo_class = 0; demo_class < num_classes_; demo_class++)
    {
	landscape.resetStage(demo_class);
	PackedIndividual ind = landscape.getNextInd(demo_class);

	while(ind.cl != -1)
	{
	    landscape.advanceStagePtr(demo_class);

	    CudaIndividual cuda_ind = { ind.id, ind.mid, ind.pid, ind.cl, ind.gen, ind.changed, ind.lastrep };
	    individuals.push_back(cuda_ind);
	    
	    ind = landscape.getNextInd(demo_class);
	}
    }
}

void Landscape_gpu::InjectIndividualsIntoLandscape(const std::list<CudaIndividual> &individuals, Landscape_statistics &landscape)
{
    std::vector<int> pop_sizes(num_classes_);

    for(int i = 0; i < num_classes_; i++)
    {
	pop_sizes[i] = CountIndividualsInClass(i);
    }

    landscape.popsizeset(pop_sizes);

    landscape.GCAlleles();

    StepAlleleTbl *s = new StepAlleleTbl;
    landscape.Atbl_push_back(s);

    std::list<CudaIndividual>::const_iterator iter = individuals.begin();

    while(iter != individuals.end())
    {
	const CudaIndividual &cuda_ind = *iter;

	assert(cuda_ind.cl >= 0);
	assert(cuda_ind.cl < num_classes_);

	PackedIndividual ind;

	ind.id = cuda_ind.id;
	ind.mid = cuda_ind.mid;
	ind.pid = cuda_ind.pid;
	ind.cl = cuda_ind.cl;
	ind.gen = cuda_ind.gen;
	ind.changed = cuda_ind.changed;
	ind.lastrep = cuda_ind.lastrep;

	std::cout << ind << std::endl;

	landscape.addIndividual(ind, -1);

	iter++;
    }
}

// Getters/Setters

void Landscape_gpu::SetIndividuals(const std::vector<CudaIndividual> &h_individuals)
{
//    printf("SetIndividuals():\n");

    thrust::fill(classes_.begin(), classes_.end(), -1);
    int num_to_transfer = h_individuals.size();
    int num_blocks = 0;

//    printf("Transferring %d individuals to GPU\n", num_to_transfer);

    if(num_to_transfer > MAX_INDIVIDUAL_TRANSFER_SIZE)
    {
//	printf("Large number of individuals to transfer.\n");
//	printf("Partitioning individuals into smaller blocks of size %d.\n",
//	    MAX_INDIVIDUAL_TRANSFER_SIZE);

	thrust::device_vector<CudaIndividual> d_individuals(MAX_INDIVIDUAL_TRANSFER_SIZE);
	int num_transferred = 0;

	while(num_transferred < num_to_transfer)
	{
	    num_blocks++;

	    int transfer_size = num_to_transfer > MAX_INDIVIDUAL_TRANSFER_SIZE ? MAX_INDIVIDUAL_TRANSFER_SIZE : num_to_transfer;
	    d_individuals.assign(h_individuals.begin() + num_transferred, h_individuals.begin() + num_transferred + transfer_size);

//	    printf("Block #%d: size=%d\n", num_blocks, transfer_size);
	    
	    thrust::transform(
		d_individuals.begin(), 
		d_individuals.begin() + transfer_size,
		thrust::make_zip_iterator(
		    thrust::make_tuple(
			ids_.begin() + num_transferred,
			maternal_ids_.begin() + num_transferred,
			paternal_ids_.begin() + num_transferred,
			classes_.begin() + num_transferred,
			generations_born_.begin() + num_transferred,
			generations_last_changed_.begin() + num_transferred,
			generations_last_reproduced_.begin() + num_transferred)),
		functors::individidual_to_tuple());
	}
    }
    else
    {
	thrust::device_vector<CudaIndividual> d_individuals(h_individuals.begin(), h_individuals.begin() + num_to_transfer);
	
	thrust::transform(
	    d_individuals.begin(), 
	    d_individuals.end(), 
	    thrust::make_zip_iterator(
		thrust::make_tuple(
		    ids_.begin(),
		    maternal_ids_.begin(),
		    paternal_ids_.begin(),
		    classes_.begin(),
		    generations_born_.begin(),
		    generations_last_changed_.begin(),
		    generations_last_reproduced_.begin())),
	    functors::individidual_to_tuple());
    }

    CompactIndividuals();
}

void Landscape_gpu::GetIndividuals(std::list<CudaIndividual> &h_individuals)
{
    const int pop_size = CountIndividuals();

    thrust::device_vector<CudaIndividual> d_individuals(pop_size);

    thrust::transform(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		ids_.begin(),
		maternal_ids_.begin(),
		paternal_ids_.begin(),
		classes_.begin(),
		generations_born_.begin(),
		generations_last_changed_.begin(),
		generations_last_reproduced_.begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		ids_.begin() + pop_size,
		maternal_ids_.begin() + pop_size,
		paternal_ids_.begin() + pop_size,
		classes_.begin() + pop_size,
		generations_born_.begin() + pop_size,
		generations_last_changed_.begin() + pop_size,
		generations_last_reproduced_.begin() + pop_size)),
	d_individuals.begin(),
	functors::tuple_to_individual());

    thrust::host_vector<CudaIndividual> th_individuals(pop_size);
    thrust::copy(d_individuals.begin(), d_individuals.end(), th_individuals.begin());

    h_individuals.assign(th_individuals.begin(), th_individuals.end());
}

// Private Simulation Methods

int Landscape_gpu::CountIndividuals()
{
    return thrust::count_if(classes_.begin(), classes_.end(), functors::non_negative());
}

int Landscape_gpu::CountIndividualsInHabitat(int habitat)
{
    assert(habitat >= 0);
    assert(habitat < num_habitats_);

    int min_class = habitat * num_stages_;
    int max_class = min_class + num_stages_;

    return thrust::count_if(classes_.begin(), classes_.end(), functors::in_range(min_class, max_class));
}

int Landscape_gpu::CountIndividualsInClass(int cl)
{
    assert(cl >= 0);
    assert(cl < num_classes_);

    return thrust::count(classes_.begin(), classes_.end(), cl);
}

void Landscape_gpu::CalculateMaleGameteMatrix()
{
    std::vector<double> n(num_classes_ * num_classes_);
    std::vector<int> class_sizes(num_classes_);

    thrust::host_vector<double> M(num_classes_ * num_classes_);

    for(int cl = 0; cl < num_classes_; cl++)
    {
	class_sizes[cl] = CountIndividualsInClass(cl);
    }

    for(int to = 0; to < num_classes_; to++)
    {
	int column_start_index = to * num_classes_;
	double sum_weighted_class_size = 0.;

	for(int from = 0; from < num_classes_; from++)
	{
	    int linear_index = column_start_index + from;

	    int m_value = host_M_[linear_index];
	    int class_size = double(class_sizes[from]);

	    n[linear_index] = (m_value * class_size);
	    sum_weighted_class_size += n[linear_index];
	}

	if(sum_weighted_class_size > 0)
	{
	    double sum_probability = 0.;

	    for(int from = 0; from < num_classes_; from++)
	    {
		int linear_index = column_start_index + from;

		double probability = n[linear_index] / sum_weighted_class_size;

		M[linear_index] = probability;
		sum_probability =+ probability;
	    }

	    if (sum_probability > 1.)
	    {
		if (sum_probability > 1.1) //something is very wacky and the program should terminate
		{
		    cerr << "The probabilities of choosing a male gamete class total to more than 1: total = "<< sum_probability << endl;
		    assert(sum_probability <= 1);
		}
		else
		{
		    for(int from = 0; from < num_classes_; from++)
		    {
			int linear_index = column_start_index + from;
			M[linear_index] = M[linear_index] / sum_probability;
		    }
		}
	    }
	}
    }

    M_ = M;
}

void Landscape_gpu::SetTransitionMatrices()
{
    S_ = host_S_;
    R_ = host_R_;
}

void Landscape_gpu::CompactIndividuals()
{
    const int pop_size_before = CountIndividuals();

    thrust::copy_if(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		ids_.begin(),
		maternal_ids_.begin(),
		paternal_ids_.begin(),
		classes_.begin(),
		generations_born_.begin(),
		generations_last_changed_.begin(),
		generations_last_reproduced_.begin(),
		genotypes_.begin())), 
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		ids_.end(),
		maternal_ids_.end(),
		paternal_ids_.end(),
		classes_.end(),
		generations_born_.end(),
		generations_last_changed_.end(),
		generations_last_reproduced_.end(),
		genotypes_.end())),
	classes_.begin(),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		ids_.begin(),
		maternal_ids_.begin(),
		paternal_ids_.begin(),
		classes_.begin(),
		generations_born_.begin(),
		generations_last_changed_.begin(),
		generations_last_reproduced_.begin(),
		genotypes_.begin())),
	functors::non_negative());

    thrust::fill(classes_.begin() + pop_size_before, classes_.end(), -1);

    assert(pop_size_before == CountIndividuals());
}

// Debug Methods

void Landscape_gpu::PrintSimulationStatistics()
{
    timer_.PrintStatistics();
}

void Landscape_gpu::PrintIndividuals(const int num_to_print)
{
    kernel_params_.num_individuals = (num_to_print < max_size_) ? num_to_print : max_size_;
    cuda::PrintIndividuals(kernel_params_);
}

void Landscape_gpu::PrintLandscape()
{
    PrintIndividuals(max_size_);
}

} /* namespace rmetasim_gpu */

/*
  ;;; Local Variables:        ***
  ;;; mode: C++               ***
  ;;; minor-mode:  font-lock  ***
  ;;; End:                    ***
*/
