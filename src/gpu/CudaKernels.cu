#include "hip/hip_runtime.h"
/*
 * Copyright 2012 Thomas Fussell
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */


/*! \file CudaKernels.cu
 *  \brief Defines several simulation function interfaces as well as associated
 *         lower-level CUDA kernel calls.
 */

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <cassert>
#include <cstdlib>
#include <cstdio>
#include <cmath>

#include "gpu/CudaKernels.cuh"
#include "const.h"
#include "gpu/const_gpu.h"

namespace rmetasim_gpu {
namespace cuda {

// Macros
#define CudaCall(X) CudaCall_((X), __FUNCTION__, __FILE__, __LINE__)

// Utility Functions

/*! \p CudaCall_ is a debugging function that is used both before and after 
 *  kernel calls to ensure no errors occured. This is not used directly but
 *  rather by the macro \p CudaCall which gives error location information.
 *
 *  \param error The CUDA error upon which this function is called
 *  \param function A string holding the name of the calling function.
 *  \param file The file in which this error was generated.
 *  \param line The line in the file at which this error ocurred.
 */
__host__
void CudaCall_(hipError_t error, const char *function, const char *file, int line)
{
    if(error != hipSuccess)
    {
	const char *errorStr = hipGetErrorString(error);
	debug_printf("CUDA Error in %s (%s:%d): %s\n", function, file, line, errorStr);
    }

    return;
}

/*! \p CalculateNumBlocks is called before each kernel invocation to calculate
 *  the upper bound of needed blocks for a given number of elements and threads.
 *
 *  \param num_items Number of items to be operated upon
 *  \param num_threads Number of threads allowed per block, multiple of 32
 *  \return The result of (int)ceil((float)num_items/num_threads)
 */
__host__
int CalculateNumBlocks(const int num_items, const int num_threads)
{
    return (num_items - 1) / num_threads + 1;
}

/*! \p log_gamma calculates the natural logarithm of the gamma function on x.
 *  This function can only be called from device contexts.
 *
 *  \param xx The value to be used in the gamma function
 *  \return The result ln(gamma(x))
 */
__device__ 
float log_gamma(const float xx) 
{
    int j;
    float x,tmp,y,ser;

    const float cof[14] = { 57.1562356658629235,     -59.5979603554754912,
			    14.1360979747417471,     -0.491913816097620199,
			    .339946499848118887e-4,  .465236289270485756e-4,
			    -.983744753048795646e-4, .158088703224912494e-3,
			    -.210264441724104883e-3, .217439618115212643e-3,
			    -.164318106536763890e-3, .844182239838527433e-4,
			    -.261908384015814087e-4, .368991826595316234e-5 };
    
    y = x = xx;
    tmp = x + 5.24218750000000000;
    tmp = (x + 0.5) * log(tmp) - tmp;
    ser = 0.999999999999997092;

    for(j = 0; j < 14; j++) 
    {
	ser += cof[j] / ++y;
    }

    return tmp + log(2.5066282746310005 * ser / x);
}


/*! \p rpoisson generates a new random value from the poisson distribution of a
 *  given lambda. This function can only be called from device contexts.
 *
 *  \param randState A pointer to an initilized CUDA hiprandState
 *  \param lambda The lambda value of the poisson distribution
 *  \return The Poisson random variable
 */
__device__ 
int rpoisson(hiprandState *randState, const float lambda) 
{
    float u,u2,v,v2,p,t,lfac,lamexp,sqlam,loglam;
    int k;
    if(lambda < 5.) 
    {
	lamexp = exp(-lambda);
	k = -1;
	t = 1.;

	do 
	{
	    ++k;
	    t *= hiprand_uniform(randState);
	} while (t > lamexp);
    } 
    else 
    {
	sqlam = sqrt(lambda);
	loglam = log(lambda);

	for(;;) 
	{
	    u = 0.64 * hiprand_uniform(randState);
	    v = -0.68 + 1.28 * hiprand_uniform(randState);

	    if(lambda > 13.5) 
	    {
		v2 = v * v;
		if(v >= 0.) 
		{ 
		    if(v2 > 6.5 * u * (0.64 - u) * (u + 0.2)) 
		    {
			continue; 
		    }
		}
		else 
		{
		    if(v2 > 9.6 * u * (0.66 - u) * (u + 0.07)) 
		    {
			continue;
		    }
		}
	    }

	    k = int(floor(sqlam * (v / u) + lambda + 0.5));

	    if(k < 0)
	    {
		continue;
	    }

	    u2 = u * u;

	    if(lambda > 13.5) 
	    {
		if(v >= 0.)
		{
		    if(v2 < 15.2 * u2 * (0.61 - u) * (0.8 - u))
		    {
			break;
		    }
		}
		else 
		{
		    if (v2 < 6.76 * u2 * (0.62 - u) * (1.4 - u)) 
		    {
			break;
		    }
		}
	    }

	    lfac = log_gamma(k + 1.);
	    p = sqlam * exp(-lambda + k * loglam - lfac);

	    if (u2 < p)
	    { 
		break;
	    }
	}
    }

    return k;
}


// Debugging Kernels

__global__
void PrintIndividualsKernel(const KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;
	
    if(global_index < params.num_individuals)
    {
	printf("[%d] : id=%d, mid=%d, pid=%d, cl=%d, born=%d, changed=%d lastrep=%d\n", global_index, 
	    params.ids[global_index], params.maternal_ids[global_index], params.paternal_ids[global_index], 
	    params.classes[global_index], params.generations_born[global_index], 
	    params.generations_last_changed[global_index], params.generations_last_reproduced[global_index]);
    }
}

void PrintIndividuals(KernelParameters &params)
{
    if(params.num_individuals > 0)
    {
	const int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	CudaCall(hipGetLastError());

	PrintIndividualsKernel<<< num_blocks, num_threads >>>(params);
	
	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}


// Initialization Kernels

__global__ 
void InitializeRandStatesKernel(const unsigned long *seeds, KernelParameters params)
{
    const int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    if(global_index < params.max_individuals)
    {
	unsigned long seed = seeds[global_index];
	hiprandState *rand_state = params.rand_states + global_index;
	hiprand_init(seed, 0, 0, rand_state);
    }
}

void InitializeRandStates(const unsigned long *seeds, KernelParameters &params)
{
    if(params.max_individuals > 0)
    {
	const unsigned int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const unsigned int num_blocks = CalculateNumBlocks(params.max_individuals, num_threads);

	CudaCall(hipGetLastError());

	InitializeRandStatesKernel<<< num_blocks, num_threads >>>(seeds, params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());  
    }
}


// Survival Kernels

__global__ 
void SurviveKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(global_index < params.num_individuals)
    {
	const short old_class = params.classes[global_index];

	if(old_class >= 0)
	{
	    short new_class = -1;
	    short start_index = old_class * params.num_classes;

	    float cumulative_prob = 0;
	    float rand = hiprand_uniform(params.rand_states + global_index);

	    for(int j = 0; j < params.num_classes; j++)
	    {
		cumulative_prob += params.S[start_index + j];

		if(new_class == -1 && rand < cumulative_prob)
		{
		    new_class = j;
		}
	    }

	    params.classes[global_index] = new_class;

	    if(new_class != old_class)
	    {
		params.generations_last_changed[global_index] = params.current_generation;
	    }
	}
    }
}

void Survive(KernelParameters &params)
{	
    if(params.num_individuals > 0)
    {
	const unsigned int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const unsigned int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	SurviveKernel<<< num_blocks, num_threads >>>(params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}

__global__ 
void SurviveOffspringKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(global_index < params.num_individuals)
    {
	const short old_class = params.offspring_classes[global_index];

	if(old_class >= 0)
	{
	    short new_class = -1;
	    short start_index = old_class * params.num_classes;

	    float cumulative_prob = 0;
	    float rand = hiprand_uniform(params.rand_states + global_index);

	    for(int j = 0; j < params.num_classes; j++)
	    {
		cumulative_prob += params.S[start_index + j];

		if(new_class == -1 && rand < cumulative_prob)
		{
		    new_class = j;
		}
	    }

	    params.offspring_classes[global_index] = new_class;
	}
    }
}

void SurviveOffspring(KernelParameters &params)
{	
    if(params.num_individuals > 0)
    {	
	assert(params.num_individuals <= OFFSPRING_ALLOC_CHUNK_SIZE);
	
	const unsigned int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const unsigned int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	SurviveOffspringKernel<<< num_blocks, num_threads >>>(params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}


// Reproduction Kernels

__global__
void CalculateRandomNumberOfOffspringKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(global_index < params.num_individuals)
    {
	int from = params.classes[global_index];

	if(from >= 0)
	{
	    float mu = params.R[from * params.num_classes + params.to_state];
	    int poisson = rpoisson(params.rand_states + global_index, mu);

	    params.num_offspring[global_index] = poisson;
	}
    }
}

void CalculateRandomNumberOfOffspring(KernelParameters &params)
{
    if(params.num_individuals > 0)
    {
	const int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	CalculateRandomNumberOfOffspringKernel<<< num_blocks, num_threads >>>(params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}

__global__
void SexualReproductionKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(global_index < params.num_individuals)
    {
	short new_class = -1;
	short start_index = params.to_state * params.num_classes;
	
	float rand = hiprand_uniform(params.rand_states + global_index);
	float cumulative_prob = 0;

	for(int j = 0; j < params.num_classes; j++)
	{
	    cumulative_prob += params.S[start_index + j];
	    
	    if(new_class == -1 && rand < cumulative_prob)
	    {
		new_class = j;
	    }
	}

	params.offspring_classes[global_index] = params.to_state;
	    
	int maternal_index = params.offspring_maternal_indices[global_index];
	int paternal_index = -1;
	
	if(params.multiple_paternity)
	{
	    paternal_index = params.offspring_paternal_indices[global_index];
	}
	else
	{
	    paternal_index = params.offspring_paternal_indices[maternal_index];
	}

	if(maternal_index != -1 && paternal_index != -1)
	{
	    rand = hiprand_uniform(params.rand_states + global_index);

	    if(rand < params.selfing_rate)
	    {
		paternal_index = params.offspring_maternal_indices[global_index];
	    }
	    
	    params.maternal_ids[global_index] = params.ids[maternal_index];
	    params.paternal_ids[global_index] = params.ids[paternal_index];
	    
	    int *mother_start = params.genotypes + maternal_index;
	    int *father_start = params.genotypes + paternal_index;
	    
	    int *self_start = params.offspring_genotypes + global_index;
	    
	    for(int j = 0; j < MAXLOCI; j++)
	    {
		char which_parent_allele = hiprand_uniform(params.rand_states + global_index);
		
		// Mother's Gamete
		self_start[j] = (!!(which_parent_allele & 1)) * mother_start[j] + 
		    (1 - !!(which_parent_allele & 1)) * mother_start[j + 1];
		
		// Father's Gamete
		self_start[j + 1] = (!!(which_parent_allele & 2)) * father_start[j] + 
		    (1 - !!(which_parent_allele & 2)) * father_start[j + 1];
	    }
	}
	else
	{
	    printf("Error in SexualReproductionKernel(): no parent found for offspring\n");
	}
    }
}

void SexualReproduction(KernelParameters &params)
{
    if(params.num_individuals > 0)
    {
	const int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	CudaCall(hipGetLastError());

	SexualReproductionKernel<<< num_blocks, num_threads >>>(params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}

__global__
void FindMatesKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;

    if(global_index < params.num_individuals)
    {
	if(params.multiple_paternity || 
	    (!params.multiple_paternity && (params.num_offspring[global_index] >= 0)))
	{
	    const short from = params.classes[global_index];

	    short target_father_class = -1;
	    float cumulative_prob = 0;
	    float rand = hiprand_uniform(params.rand_states + global_index);
	    
	    for(int to = 0; to < params.num_classes; to++)
	    {
		float m_value = params.M[from + to * params.num_classes];
		cumulative_prob += m_value;
		
		if(target_father_class == -1 && rand < cumulative_prob)
		{
		    target_father_class = to;
		}
	    }

	    //printf("FindMatesKernel: thread=%d, from=%d, target=%d\n",global_index,from,target_father_class);

	    if(target_father_class != -1)
	    {
		int father_index = hiprand(params.rand_states + global_index) * params.num_individuals - 1;

		short father_class = params.classes[father_index];
		short num_tries = 0;
		
		while(num_tries++ < MAX_MATE_SEARCH_ATTEMPTS && target_father_class != father_class)
		{
		    father_index = hiprand(params.rand_states + global_index) * params.num_individuals - 1;
		    father_class = params.classes[father_index];
		}
		
		if(father_class == target_father_class)
		{
		    params.offspring_paternal_indices[global_index] = father_index;
		}
		else
		{
		    // No father in target class found, what should we do?
		    //assert(father_class == target_father_class);
		} 
	    }
	}
    }
}

void FindMates(KernelParameters &params)
{
    if(params.num_individuals > 0)
    {
	const int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	//printf("cuda::FindMates(): num_threads=%d, num_blocks=%d\n",num_threads,num_blocks);

	CudaCall(hipGetLastError());

	FindMatesKernel<<< num_blocks, num_threads >>>(params);

	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}


// Miscellaneous Kernels

// Note: This function will freeze program execution if there are not enough open positions.
__global__
void FillShuffleVectorKernel(KernelParameters params)
{
    const int global_index = blockDim.x * blockIdx.x + threadIdx.x;
    const int emptiness_indicator = -1;

    if(global_index < params.num_individuals
	&& params.classes[global_index] == params.state)
    {
	hiprandState *rand_state = params.rand_states + global_index;

	int new_location = hiprand(rand_state) % params.max_individuals;
	bool succeeded = atomicCAS(params.shuffling_dartboard + new_location, 
	    emptiness_indicator, global_index) == emptiness_indicator;

	while(!succeeded)
	{
	    new_location = hiprand(rand_state) % params.max_individuals;
	    succeeded = atomicCAS(params.shuffling_dartboard + new_location, 
		emptiness_indicator, global_index) == emptiness_indicator;
	}
    }
}

void FillShuffleVector(KernelParameters &params)
{
    if(params.num_individuals > 0)
    {
	const int num_threads = MAX_CUDA_THREADS_PER_BLOCK;
	const int num_blocks = CalculateNumBlocks(params.num_individuals, num_threads);

	CudaCall(hipGetLastError());

	FillShuffleVectorKernel<<< num_blocks, num_threads >>>(params);
	
	CudaCall(hipGetLastError());
	CudaCall(hipDeviceSynchronize());
    }
}

} // end cuda
} // end rmetasim_gpu

/*
  ;;; Local Variables:        ***
  ;;; mode: C++               ***
  ;;; minor-mode:  font-lock  ***
  ;;; End:                    ***
*/
